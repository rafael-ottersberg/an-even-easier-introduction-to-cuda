
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1<<20;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    int blockSize = 1024;
    int numOfBlocks = (N + blockSize - 1) / blockSize;

    add<<<numOfBlocks,blockSize>>>(N, x, y);

    hipDeviceSynchronize();
    
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
        std::cout << "maxError: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}

